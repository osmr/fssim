#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void multiply_kernel(float* output, const float* input, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = input[idx] * 2.0f;
    }
}

torch::Tensor multiply_by_two_cuda(torch::Tensor input) {
    TORCH_CHECK(input.device().is_cuda(), "Tensor must be on CUDA device");
    TORCH_CHECK(input.dtype() == torch::kFloat32, "Only float32 is supported");

    torch::Tensor output = torch::empty_like(input);

    int size = output.numel();
    int threads = 256;
    int blocks = (size + threads - 1) / threads;

    multiply_kernel<<<blocks, threads>>>(
        output.data_ptr<float>(),
        input.data_ptr<float>(),
        size
    );

    hipError_t err = hipDeviceSynchronize();
    TORCH_CHECK(err == hipSuccess, "CUDA error: ", hipGetErrorString(err));

    return output;
}
