#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_cooperative_groups.h>
#include <algorithm>
#include <iostream>
#include <c10/cuda/CUDAGuard.h>

namespace cg = cooperative_groups;


// ------------------------------------------
// Constant Memory for Gaussian Coefficients
// ------------------------------------------
__constant__ float gauss_coefs[11] = {
    0.001028380123898387f,
    0.0075987582094967365f,
    0.036000773310661316f,
    0.10936068743467331f,
    0.21300552785396576f,
    0.26601171493530273f,
    0.21300552785396576f,
    0.10936068743467331f,
    0.036000773310661316f,
    0.0075987582094967365f,
    0.001028380123898387f
};

// ------------------------------------------
// Block and Shared Memory Dimensions
// ------------------------------------------
#define BLOCK_X 16
#define BLOCK_Y 16
#define HALO    5

#define SHARED_X (BLOCK_X + 2 * HALO)
#define SHARED_Y (BLOCK_Y + 2 * HALO)

// For partial results after horizontal pass
#define CONV_X BLOCK_X
#define CONV_Y SHARED_Y

// ------------------------------------------
// Utility: Safe pixel fetch w/ zero padding
// ------------------------------------------
__device__ __forceinline__ float get_pix_value(const float* img, const int b, const int c, const int y, const int x,
                                               const int CH, const int H, const int W) {
    if (x < 0 || x >= W || y < 0 || y >= H) {
        return 0.0f;
    }
    return img[b * CH * H * W + c * H * W + y * W + x];
}


//-------------------------------------------------------------------------------------------


/**
 * @brief Forward Kernel: Fused SSIM.
 *
 * Fused SSIM Map Calculation (CUDA Kernel)
 *  - Two-pass convolution to get mu1, mu2, sigma1_sq, sigma2_sq, sigma12, etc.
 *  - Writes final SSIM map to ssim_map
 *  - Optionally writes partial derivatives to dm_dmu1, dm_dsigma1_sq, dm_dsigma12
 */
__global__ void ssim_kernel(const int H,
                            const int W,
                            const int CH,
                            const float C1,
                            const float C2,
                            const float* __restrict__ img1,
                            const float* __restrict__ img2,
                            float* __restrict__ ssim_map,
                            float* __restrict__ dm_dmu1,
                            float* __restrict__ dm_dsigma1_sq,
                            float* __restrict__ dm_dsigma12) {
    auto block = cg::this_thread_block();
    const int bIdx   = block.group_index().z;  // batch index
    const int pix_y  = block.group_index().y * BLOCK_Y + block.thread_index().y;
    const int pix_x  = block.group_index().x * BLOCK_X + block.thread_index().x;
    const int pix_id = pix_y * W + pix_x;
    const int num_pix = H * W;

    // Shared memory for the tile (img1, img2)
    __shared__ float s_tile[SHARED_Y][SHARED_X][2];
    // After horizontal pass, store partial sums here
    // x_conv[y][x] -> (sum_x, sum_x^2, sum_y, sum_y^2, sum_xY)
    __shared__ float x_conv[CONV_Y][CONV_X][5];

    // Each block processes B x C sub-batches. We loop over channels:
    for (int c = 0; c < CH; ++c) {
        // ------------------------------------------------------------
        // 1) Load (img1, img2) tile + halo into shared memory
        // ------------------------------------------------------------
        {
            const int tile_size = SHARED_Y * SHARED_X;
            const int threads = BLOCK_X * BLOCK_Y;
            const int steps = (tile_size + threads - 1) / threads;

            const int tile_start_y = block.group_index().y * BLOCK_Y;
            const int tile_start_x = block.group_index().x * BLOCK_X;

            for (int s = 0; s < steps; ++s) {
                int tid = s * threads + block.thread_rank();
                if (tid < tile_size) {
                    const int local_y = tid / SHARED_X;
                    const int local_x = tid % SHARED_X;
                    const int gy = tile_start_y + local_y - HALO;
                    const int gx = tile_start_x + local_x - HALO;

                    const float X = get_pix_value(img1, bIdx, c, gy, gx, CH, H, W);
                    const float Y = get_pix_value(img2, bIdx, c, gy, gx, CH, H, W);

                    s_tile[local_y][local_x][0] = X;
                    s_tile[local_y][local_x][1] = Y;
                }
            }
        }
        block.sync();

        // ------------------------------------------------------------
        // 2) Horizontal convolution (11x1) in shared memory
        //    We'll accumulate symmetrical pairs around center.
        // ------------------------------------------------------------
        {
            const int ly = threadIdx.y;
            const int lx = threadIdx.x + HALO;  // skip left halo

            float sum_x   = 0.f;
            float sum_x2  = 0.f;
            float sum_y   = 0.f;
            float sum_y2  = 0.f;
            float sum_xY  = 0.f;

            // #pragma unroll for those 5 pairs
            #pragma unroll
            for (int d = 1; d <= HALO; ++d) {
                const float w = gauss_coefs[HALO - d];
                const float x_left  = s_tile[ly][lx - d][0];
                const float y_left  = s_tile[ly][lx - d][1];
                const float x_right = s_tile[ly][lx + d][0];
                const float y_right = s_tile[ly][lx + d][1];

                sum_x  += (x_left + x_right) * w;
                sum_x2 += ((x_left * x_left) + (x_right * x_right)) * w;
                sum_y  += (y_left + y_right) * w;
                sum_y2 += ((y_left * y_left) + (y_right * y_right)) * w;
                sum_xY += ((x_left * y_left) + (x_right * y_right)) * w;
            }
            // center
            {
                const float center_x = s_tile[ly][lx][0];
                const float center_y = s_tile[ly][lx][1];
                const float wc = gauss_coefs[HALO];
                sum_x  += center_x * wc;
                sum_x2 += (center_x * center_x) * wc;
                sum_y  += center_y * wc;
                sum_y2 += (center_y * center_y) * wc;
                sum_xY += (center_x * center_y) * wc;
            }

            // Write out partial sums
            x_conv[ly][threadIdx.x][0] = sum_x;
            x_conv[ly][threadIdx.x][1] = sum_x2;
            x_conv[ly][threadIdx.x][2] = sum_y;
            x_conv[ly][threadIdx.x][3] = sum_y2;
            x_conv[ly][threadIdx.x][4] = sum_xY;

            // Possibly handle second row in same warp
            const int ly2 = ly + BLOCK_Y;
            if (ly2 < CONV_Y) {
                sum_x   = 0.f; sum_x2  = 0.f;
                sum_y   = 0.f; sum_y2  = 0.f;
                sum_xY  = 0.f;

                #pragma unroll
                for (int d = 1; d <= HALO; ++d) {
                    const float w = gauss_coefs[HALO - d];
                    const float x_left  = s_tile[ly2][lx - d][0];
                    const float y_left  = s_tile[ly2][lx - d][1];
                    const float x_right = s_tile[ly2][lx + d][0];
                    const float y_right = s_tile[ly2][lx + d][1];

                    sum_x  += (x_left + x_right) * w;
                    sum_x2 += ((x_left * x_left) + (x_right * x_right)) * w;
                    sum_y  += (y_left + y_right) * w;
                    sum_y2 += ((y_left * y_left) + (y_right * y_right)) * w;
                    sum_xY += ((x_left * y_left) + (x_right * y_right)) * w;
                }
                // center
                {
                    const float cx = s_tile[ly2][lx][0];
                    const float cy = s_tile[ly2][lx][1];
                    const float wc = gauss_coefs[HALO];
                    sum_x  += cx * wc;
                    sum_x2 += (cx * cx) * wc;
                    sum_y  += cy * wc;
                    sum_y2 += (cy * cy) * wc;
                    sum_xY += (cx * cy) * wc;
                }
                x_conv[ly2][threadIdx.x][0] = sum_x;
                x_conv[ly2][threadIdx.x][1] = sum_x2;
                x_conv[ly2][threadIdx.x][2] = sum_y;
                x_conv[ly2][threadIdx.x][3] = sum_y2;
                x_conv[ly2][threadIdx.x][4] = sum_xY;
            }
        }
        block.sync();

        // ------------------------------------------------------------
        // 3) Vertical convolution (1x11) + final SSIM
        // ------------------------------------------------------------
        {
            const int ly = threadIdx.y + HALO;
            const int lx = threadIdx.x;

            float out0 = 0.f, out1 = 0.f, out2 = 0.f, out3 = 0.f, out4 = 0.f;

            #pragma unroll
            for (int d = 1; d <= HALO; ++d) {
                const float w = gauss_coefs[HALO - d];
                const float* top = x_conv[ly - d][lx];
                const float* bot = x_conv[ly + d][lx];

                out0 += (top[0] + bot[0]) * w;
                out1 += (top[1] + bot[1]) * w;
                out2 += (top[2] + bot[2]) * w;
                out3 += (top[3] + bot[3]) * w;
                out4 += (top[4] + bot[4]) * w;
            }
            // center
            {
                const float wC = gauss_coefs[HALO];
                const float* ctr = x_conv[ly][lx];
                out0 += ctr[0] * wC;
                out1 += ctr[1] * wC;
                out2 += ctr[2] * wC;
                out3 += ctr[3] * wC;
                out4 += ctr[4] * wC;
            }

            if (pix_x < W && pix_y < H) {
                const float mu1 = out0;
                const float mu2 = out2;
                const float mu1_sq = mu1 * mu1;
                const float mu2_sq = mu2 * mu2;

                const float sigma1_sq = out1 - mu1_sq;
                const float sigma2_sq = out3 - mu2_sq;
                const float sigma12   = out4 - mu1 * mu2;

                const float A = mu1_sq + mu2_sq + C1;
                const float B = sigma1_sq + sigma2_sq + C2;
                const float C_ = 2.f * mu1 * mu2 + C1;
                const float D_ = 2.f * sigma12 + C2;

                const float val = (C_ * D_) / (A * B);

                const int global_idx = bIdx * CH * num_pix + c * num_pix + pix_id;
                ssim_map[global_idx] = val;

                if (dm_dmu1) {
                    // partial derivatives
                    const float d_m_dmu1 = (
                        (mu2 * 2.f * D_) / (A * B)
                        - (mu2 * 2.f * C_) / (A * B)
                        - (mu1 * 2.f * C_ * D_) / (A * A * B)
                        + (mu1 * 2.f * C_ * D_) / (A * B * B)
                    );
                    const float d_m_dsigma1_sq = (-C_ * D_) / (A * B * B);
                    const float d_m_dsigma12   = (2.f * C_) / (A * B);

                    dm_dmu1[global_idx]       = d_m_dmu1;
                    dm_dsigma1_sq[global_idx] = d_m_dsigma1_sq;
                    dm_dsigma12[global_idx]   = d_m_dsigma12;
                }
            }
        }
    }
}

/**
 * @brief PyTorch Interface (Forward).
 *
 * PyTorch Interface for SSIM Map calculation (Forward pass)
 *   Returns (ssim_map, dm_dmu1, dm_dsigma1_sq, dm_dsigma12).
 *   If train=false, derivative Tensors are empty.
 */
std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor> ssim_cuda(float C1,
                                                                                 float C2,
                                                                                 const torch::Tensor& img1,
                                                                                 const torch::Tensor& img2,
                                                                                 const bool train) {
    TORCH_CHECK(img1.device().is_cuda(), "Tensor img1 must be on CUDA device");
    TORCH_CHECK(img2.device().is_cuda(), "Tensor img2 must be on CUDA device");
    TORCH_CHECK(img1.get_device() == img2.get_device(), "Input tensors must be on the same device");

    const at::cuda::OptionalCUDAGuard device_guard(device_of(img1));
    const int B  = img1.size(0);
    const int CH = img1.size(1);
    const int H  = img1.size(2);
    const int W  = img1.size(3);

    // Launch config
    const dim3 grid((W + BLOCK_X - 1) / BLOCK_X,
                    (H + BLOCK_Y - 1) / BLOCK_Y,
                    B);
    const dim3 block(BLOCK_X, BLOCK_Y);

    // Output SSIM map
    auto ssim_map = torch::zeros_like(img1, img1.options()).contiguous();

    // Optionally allocate derivative Tensors
    auto dm_dmu1       = train ? torch::zeros_like(img1) : torch::empty({0}, img1.options());
    auto dm_dsigma1_sq = train ? torch::zeros_like(img1) : torch::empty({0}, img1.options());
    auto dm_dsigma12   = train ? torch::zeros_like(img1) : torch::empty({0}, img1.options());

    ssim_kernel<<<grid, block>>>(
        H, W, CH, C1, C2,
        img1.contiguous().data_ptr<float>(),
        img2.contiguous().data_ptr<float>(),
        ssim_map.data_ptr<float>(),
        train ? dm_dmu1.data_ptr<float>()       : nullptr,
        train ? dm_dsigma1_sq.data_ptr<float>() : nullptr,
        train ? dm_dsigma12.data_ptr<float>()   : nullptr
    );

    const hipError_t err = hipDeviceSynchronize();
    TORCH_CHECK(err == hipSuccess, "CUDA error: ", hipGetErrorString(err));

    return std::make_tuple(ssim_map, dm_dmu1, dm_dsigma1_sq, dm_dsigma12);
}


//-------------------------------------------------------------------------------------------


/**
 * @brief Backward Kernel: Fused SSIM.
 *
 * Backward pass for fused SSIM Map Calculation (CUDA Kernel): Apply chain rule to get dL/d(img1) from partial
 *    derivatives (dm_dmu1, dm_dsigma1_sq, dm_dsigma12) and dL/dmap (the gradient from above).
 */
__global__ void ssim_backward_kernel(const int H,
                                     const int W,
                                     const int CH,
                                     const float C1,
                                     const float C2,
                                     const float* __restrict__ img1,
                                     const float* __restrict__ img2,
                                     const float* __restrict__ dL_dmap,
                                     float* __restrict__ dL_dimg1,
                                     const float* __restrict__ dm_dmu1,
                                     const float* __restrict__ dm_dsigma1_sq,
                                     const float* __restrict__ dm_dsigma12) {
    auto block = cg::this_thread_block();
    const int pix_y  = block.group_index().y * BLOCK_Y + block.thread_index().y;
    const int pix_x  = block.group_index().x * BLOCK_X + block.thread_index().x;
    const int pix_id = pix_y * W + pix_x;
    const int num_pix = H * W;
    const int bIdx   = block.group_index().z;

    // Shared memory for the fused data:
    // [0]: dm_dmu1*dL, [1]: dm_dsigma1_sq*dL, [2]: dm_dsigma12*dL
    __shared__ float s_data[3][SHARED_Y][SHARED_X];
    __shared__ float s_scratch[CONV_Y][CONV_X][3];

    for (int c = 0; c < CH; ++c) {
        float p1 = 0.f, p2 = 0.f;
        if (pix_x < W && pix_y < H) {
            p1 = get_pix_value(img1, bIdx, c, pix_y, pix_x, CH, H, W);
            p2 = get_pix_value(img2, bIdx, c, pix_y, pix_x, CH, H, W);
        }

        // (1) Load + fuse multiplication
        {
            const int start_y = block.group_index().y * BLOCK_Y;
            const int start_x = block.group_index().x * BLOCK_X;

            const int tid = threadIdx.y * blockDim.x + threadIdx.x;
            const int warp_id = tid / 32;
            const int lane_id = tid % 32;
            const int totalThreads = BLOCK_X * BLOCK_Y;
            const int num_warps = (totalThreads + 31) / 32;

            for (int row = warp_id; row < SHARED_Y; row += num_warps) {
                const int gy = start_y + row - HALO;
                for (int col = lane_id; col < SHARED_X; col += 32) {
                    const int gx = start_x + col - HALO;

                    const float chain = get_pix_value(dL_dmap,      bIdx, c, gy, gx, CH, H, W);
                    const float vmu   = get_pix_value(dm_dmu1,      bIdx, c, gy, gx, CH, H, W);
                    const float vs1   = get_pix_value(dm_dsigma1_sq,bIdx, c, gy, gx, CH, H, W);
                    const float vs12  = get_pix_value(dm_dsigma12,  bIdx, c, gy, gx, CH, H, W);

                    s_data[0][row][col] = vmu  * chain;
                    s_data[1][row][col] = vs1  * chain;
                    s_data[2][row][col] = vs12 * chain;
                }
            }
        }
        block.sync();

        // (2) Horizontal pass
        {
            const int ly = threadIdx.y;
            const int lx = threadIdx.x + HALO;

            for (int pass = 0; pass < 2; ++pass) {
                const int yy = ly + pass * BLOCK_Y;
                if (yy < CONV_Y) {
                    float accum0 = 0.f, accum1 = 0.f, accum2 = 0.f;

                    #pragma unroll
                    for (int d = 1; d <= HALO; ++d) {
                        const float w = gauss_coefs[HALO - d];
                        const float left0  = s_data[0][yy][lx - d];
                        const float left1  = s_data[1][yy][lx - d];
                        const float left2  = s_data[2][yy][lx - d];

                        const float right0 = s_data[0][yy][lx + d];
                        const float right1 = s_data[1][yy][lx + d];
                        const float right2 = s_data[2][yy][lx + d];

                        accum0 += (left0 + right0) * w;
                        accum1 += (left1 + right1) * w;
                        accum2 += (left2 + right2) * w;
                    }
                    // center
                    {
                        const float wc = gauss_coefs[HALO];
                        const float c0 = s_data[0][yy][lx];
                        const float c1 = s_data[1][yy][lx];
                        const float c2 = s_data[2][yy][lx];
                        accum0 += c0 * wc;
                        accum1 += c1 * wc;
                        accum2 += c2 * wc;
                    }

                    s_scratch[yy][threadIdx.x][0] = accum0;
                    s_scratch[yy][threadIdx.x][1] = accum1;
                    s_scratch[yy][threadIdx.x][2] = accum2;
                }
            }
        }
        block.sync();

        // (3) Vertical pass -> finalize dL/d(img1)
        if (pix_x < W && pix_y < H) {
            const int ly = threadIdx.y + HALO;
            const int lx = threadIdx.x;

            float sum0 = 0.f, sum1 = 0.f, sum2 = 0.f;

            #pragma unroll
            for (int d = 1; d <= HALO; ++d) {
                const float w = gauss_coefs[HALO - d];
                const float* top = s_scratch[ly - d][lx];
                const float* bot = s_scratch[ly + d][lx];

                sum0 += (top[0] + bot[0]) * w;
                sum1 += (top[1] + bot[1]) * w;
                sum2 += (top[2] + bot[2]) * w;
            }
            // center
            {
                const float wc = gauss_coefs[HALO];
                const float* ctr = s_scratch[ly][lx];
                sum0 += ctr[0] * wc;
                sum1 += ctr[1] * wc;
                sum2 += ctr[2] * wc;
            }

            // final accumulation
            const float dL_dpix = sum0 + (2.f * p1) * sum1 + (p2) * sum2;

            const int out_idx = bIdx * CH * num_pix + c * num_pix + pix_id;
            dL_dimg1[out_idx] = dL_dpix;
        }
        block.sync();
    }
}

/**
 * @brief PyTorch Interface (Backward).
 *
 * PyTorch Interface for SSIM Map calculation (Backward pass)
 *   Takes the gradient wrt the SSIM map and
 *   the partial derivatives from forward;
 *   returns dL/d(img1).
 */
torch::Tensor ssim_backward_cuda(const float C1,
                                 const float C2,
                                 const torch::Tensor& img1,
                                 const torch::Tensor& img2,
                                 const torch::Tensor& dL_dmap,
                                 const torch::Tensor& dm_dmu1,
                                 const torch::Tensor& dm_dsigma1_sq,
                                 const torch::Tensor& dm_dsigma12) {
    const at::cuda::OptionalCUDAGuard device_guard(device_of(img1));
    const int B  = img1.size(0);
    const int CH = img1.size(1);
    const int H  = img1.size(2);
    const int W  = img1.size(3);

    auto dL_dimg1 = torch::zeros_like(img1);

    const dim3 grid((W + BLOCK_X - 1) / BLOCK_X,
                    (H + BLOCK_Y - 1) / BLOCK_Y,
                    B);
    const dim3 block(BLOCK_X, BLOCK_Y);

    ssim_backward_kernel<<<grid, block>>>(
        H, W, CH, C1, C2,
        img1.contiguous().data_ptr<float>(),
        img2.contiguous().data_ptr<float>(),
        dL_dmap.contiguous().data_ptr<float>(),
        dL_dimg1.data_ptr<float>(),
        dm_dmu1.contiguous().data_ptr<float>(),
        dm_dsigma1_sq.contiguous().data_ptr<float>(),
        dm_dsigma12.contiguous().data_ptr<float>()
    );

    const hipError_t err = hipDeviceSynchronize();
    TORCH_CHECK(err == hipSuccess, "CUDA error: ", hipGetErrorString(err));

    return dL_dimg1;
}
