#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_cooperative_groups.h>
#include <algorithm>
#include <iostream>
#include <c10/cuda/CUDAGuard.h>

namespace cg = cooperative_groups;


// Block and Shared Memory Dimensions
#define BLOCK_X 16
#define BLOCK_Y 16
#define HALO    5

#define SHARED_X (BLOCK_X + 2 * HALO)
#define SHARED_Y (BLOCK_Y + 2 * HALO)

// For partial results after horizontal pass
#define CONV_X BLOCK_X
#define CONV_Y SHARED_Y

// Constant Memory for 1D Gaussian Coefficients
__constant__ float gauss_coefs[11] = {
    0.001028380123898387f,
    0.0075987582094967365f,
    0.036000773310661316f,
    0.10936068743467331f,
    0.21300552785396576f,
    0.26601171493530273f,
    0.21300552785396576f,
    0.10936068743467331f,
    0.036000773310661316f,
    0.0075987582094967365f,
    0.001028380123898387f
};

/**
 * @brief Function for safely fetching a pixel with zero-padding.
 * @param[in] image Processed image.
 * @param[in] b batch index.
 * @param[in] c cannel index.
 * @param[in] y row index.
 * @param[in] x column index.
 * @param[in] channels number of image channels.
 * @param[in] height image height.
 * @param[in] width image width.
 * @return pixel value.
 */
__device__ __forceinline__ float get_pix_value(const float* const image,
                                               const int b,
                                               const int c,
                                               const int y,
                                               const int x,
                                               const int channels,
                                               const int height,
                                               const int width) {
    if (x < 0 || x >= width || y < 0 || y >= height) {
        return 0.0f;
    }
    return image[((b * channels + c) * height + y) * width + x];
}


/**
 * @brief Calculate grid dimension.
 * @param[in] batch size of image batch.
 * @param[in] height image height.
 * @param[in] width image width.
 * @return pixel value.
 */
inline dim3 calc_grid_dim(const int batch,
                          const int height,
                          const int width) {
    return dim3(
        (width + BLOCK_X - 1) / BLOCK_X,
        (height + BLOCK_Y - 1) / BLOCK_Y,
        batch);
}

//-------------------------------------------------------------------------------------------


/**
 * @brief Forward Kernel: Fused SSIM.
 *
 * Fused SSIM Map Calculation (CUDA Kernel)
 *  - Two-pass convolution to get mu1, mu2, sigma1_sq, sigma2_sq, sigma12, etc.
 *  - Writes final SSIM map to ssim_map
 *  - Optionally writes partial derivatives to dm_dmu1, dm_dsigma1_sq, dm_dsigma12
 *
 * @param[in] channels number of image channels.
 * @param[in] height image height.
 * @param[in] width image width.
 * @param[in] C1 C1-const.
 * @param[in] C2 C2-const.
 * @param[in] img1 The first image.
 * @param[in] img2 The second image.
 * @param[out] ssim_map SSIM map/image.
 * @param[out] dm_dmu1 dm_dmu1 partial derivative image.
 * @param[out] dm_dsigma1_sq dm_dsigma1_sq partial derivative image.
 * @param[out] dm_dsigma12 dm_dsigma12 partial derivative image.
*/
__global__ void ssim_kernel(const int channels,
                            const int height,
                            const int width,
                            const float C1,
                            const float C2,
                            const float* __restrict__ img1,
                            const float* __restrict__ img2,
                            float* __restrict__ ssim_map,
                            float* __restrict__ dm_dmu1,
                            float* __restrict__ dm_dsigma1_sq,
                            float* __restrict__ dm_dsigma12) {
    const cg::thread_block block = cg::this_thread_block();
    const dim3 g_idx = block.group_index();
    const dim3 thread_idx = block.thread_index();
    const int b_idx = g_idx.z;  // batch index
    const int pix_y = g_idx.y * BLOCK_Y + thread_idx.y;
    const int pix_x = g_idx.x * BLOCK_X + thread_idx.x;
    const int pix_idx = pix_y * width + pix_x;

    const int num_pix = height * width;

    const int tile_size = SHARED_Y * SHARED_X;
    const int threads = BLOCK_X * BLOCK_Y;
    const int steps = (tile_size + threads - 1) / threads;

    // Shared memory for the tile (img1, img2)
    __shared__ float s_tile[SHARED_Y][SHARED_X][2];
    // After horizontal pass, store partial sums here
    // x_conv[y][x] -> (sum_x, sum_x^2, sum_y, sum_y^2, sum_xy)
    __shared__ float x_conv[CONV_Y][CONV_X][5];

    // Each block processes B x C sub-batches. We loop over channels:
    for (int c = 0; c < channels; ++c) {
        // (1) Load (img1, img2) tile + halo into shared memory
        {
            const int tile_start_y = g_idx.y * BLOCK_Y;
            const int tile_start_x = g_idx.x * BLOCK_X;

            for (int s = 0; s < steps; ++s) {
                const int tile_id = s * threads + block.thread_rank();
                if (tile_id < tile_size) {
                    const int local_y = tile_id / SHARED_X;
                    const int local_x = tile_id % SHARED_X;
                    const int gy = tile_start_y + local_y - HALO;
                    const int gx = tile_start_x + local_x - HALO;

                    const float X = get_pix_value(img1, b_idx, c, gy, gx, channels, height, width);
                    const float Y = get_pix_value(img2, b_idx, c, gy, gx, channels, height, width);

                    s_tile[local_y][local_x][0] = X;
                    s_tile[local_y][local_x][1] = Y;
                }
            }
        }
        block.sync();

        // (2) Horizontal convolution (11x1) in shared memory. We'll accumulate symmetrical pairs around center.
        {
            const int ly = thread_idx.y;
            const int lx = thread_idx.x + HALO;  // skip left halo

            float sum_x = 0.0f, sum_x2 = 0.0f, sum_y = 0.0f, sum_y2 = 0.0f, sum_xy = 0.0f;

            #pragma unroll
            for (int d = 1; d <= HALO; ++d) {
                const float w = gauss_coefs[HALO - d];
                const float x_left  = s_tile[ly][lx - d][0];
                const float y_left  = s_tile[ly][lx - d][1];
                const float x_right = s_tile[ly][lx + d][0];
                const float y_right = s_tile[ly][lx + d][1];

                sum_x  += (x_left + x_right) * w;
                sum_x2 += ((x_left * x_left) + (x_right * x_right)) * w;
                sum_y  += (y_left + y_right) * w;
                sum_y2 += ((y_left * y_left) + (y_right * y_right)) * w;
                sum_xy += ((x_left * y_left) + (x_right * y_right)) * w;
            }
            // center
            {
                const float center_x = s_tile[ly][lx][0];
                const float center_y = s_tile[ly][lx][1];
                const float wc = gauss_coefs[HALO];
                sum_x  += center_x * wc;
                sum_x2 += (center_x * center_x) * wc;
                sum_y  += center_y * wc;
                sum_y2 += (center_y * center_y) * wc;
                sum_xy += (center_x * center_y) * wc;
            }

            // Write out partial sums
            x_conv[ly][thread_idx.x][0] = sum_x;
            x_conv[ly][thread_idx.x][1] = sum_x2;
            x_conv[ly][thread_idx.x][2] = sum_y;
            x_conv[ly][thread_idx.x][3] = sum_y2;
            x_conv[ly][thread_idx.x][4] = sum_xy;

            // Possibly handle second row in same warp
            const int ly2 = ly + BLOCK_Y;
            if (ly2 < CONV_Y) {
                sum_x = sum_x2 = sum_y = sum_y2 = sum_xy = 0.0f;

                #pragma unroll
                for (int d = 1; d <= HALO; ++d) {
                    const float w = gauss_coefs[HALO - d];
                    const float x_left  = s_tile[ly2][lx - d][0];
                    const float y_left  = s_tile[ly2][lx - d][1];
                    const float x_right = s_tile[ly2][lx + d][0];
                    const float y_right = s_tile[ly2][lx + d][1];

                    sum_x  += (x_left + x_right) * w;
                    sum_x2 += ((x_left * x_left) + (x_right * x_right)) * w;
                    sum_y  += (y_left + y_right) * w;
                    sum_y2 += ((y_left * y_left) + (y_right * y_right)) * w;
                    sum_xy += ((x_left * y_left) + (x_right * y_right)) * w;
                }
                // center
                {
                    const float cx = s_tile[ly2][lx][0];
                    const float cy = s_tile[ly2][lx][1];
                    const float wc = gauss_coefs[HALO];
                    sum_x  += cx * wc;
                    sum_x2 += (cx * cx) * wc;
                    sum_y  += cy * wc;
                    sum_y2 += (cy * cy) * wc;
                    sum_xy += (cx * cy) * wc;
                }
                x_conv[ly2][thread_idx.x][0] = sum_x;
                x_conv[ly2][thread_idx.x][1] = sum_x2;
                x_conv[ly2][thread_idx.x][2] = sum_y;
                x_conv[ly2][thread_idx.x][3] = sum_y2;
                x_conv[ly2][thread_idx.x][4] = sum_xy;
            }
        }
        block.sync();

        // (3) Vertical convolution (1x11) + final SSIM
        {
            const int ly = thread_idx.y + HALO;
            const int lx = thread_idx.x;

            float out0 = 0.0f, out1 = 0.0f, out2 = 0.0f, out3 = 0.0f, out4 = 0.0f;

            #pragma unroll
            for (int d = 1; d <= HALO; ++d) {
                const float w = gauss_coefs[HALO - d];
                const float* top = x_conv[ly - d][lx];
                const float* bot = x_conv[ly + d][lx];

                out0 += (top[0] + bot[0]) * w;
                out1 += (top[1] + bot[1]) * w;
                out2 += (top[2] + bot[2]) * w;
                out3 += (top[3] + bot[3]) * w;
                out4 += (top[4] + bot[4]) * w;
            }
            // center
            {
                const float wC = gauss_coefs[HALO];
                const float* ctr = x_conv[ly][lx];
                out0 += ctr[0] * wC;
                out1 += ctr[1] * wC;
                out2 += ctr[2] * wC;
                out3 += ctr[3] * wC;
                out4 += ctr[4] * wC;
            }

            if (pix_x < width && pix_y < height) {
                const float mu1 = out0;
                const float mu2 = out2;
                const float mu1_sq = mu1 * mu1;
                const float mu2_sq = mu2 * mu2;

                const float sigma1_sq = out1 - mu1_sq;
                const float sigma2_sq = out3 - mu2_sq;
                const float sigma12   = out4 - mu1 * mu2;

                const float A = mu1_sq + mu2_sq + C1;
                const float B = sigma1_sq + sigma2_sq + C2;
                const float C_ = 2.0f * mu1 * mu2 + C1;
                const float D_ = 2.0f * sigma12 + C2;

                const float val = (C_ * D_) / (A * B);

                const int global_idx = b_idx * channels * num_pix + c * num_pix + pix_idx;
                ssim_map[global_idx] = val;

                if (dm_dmu1) {
                    // partial derivatives
                    const float d_m_dmu1 = 2.0f * (mu2 * (D_ - C_) + mu1 * (A - B) * val) / (A * B);
                    const float d_m_dsigma1_sq = (-C_ * D_) / (A * B * B);
                    const float d_m_dsigma12   = (2.0f * C_) / (A * B);

                    dm_dmu1[global_idx] = d_m_dmu1;
                    dm_dsigma1_sq[global_idx] = d_m_dsigma1_sq;
                    dm_dsigma12[global_idx] = d_m_dsigma12;
                }
            }
        }
    }
}

/**
 * @brief PyTorch Interface (Forward).
 *
 * PyTorch Interface for SSIM Map calculation (Forward pass)
 *   Returns (ssim_map, dm_dmu1, dm_dsigma1_sq, dm_dsigma12).
 *   If train=false, derivative Tensors are empty.
 *
 * @param[in] C1 C1-const.
 * @param[in] C2 C2-const.
 * @param[in] img1 The first image.
 * @param[in] img2 The second image.
 * @param[in] train Whether to calculate partial derivatives.
 * @return Tuple of (ssim_map, dm_dmu1, dm_dsigma1_sq, dm_dsigma12).
 */
std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor> ssim_cuda(const float C1,
                                                                                 const float C2,
                                                                                 const torch::Tensor& img1,
                                                                                 const torch::Tensor& img2,
                                                                                 const bool train) {
    TORCH_CHECK(img1.device().is_cuda(), "Tensor img1 must be on CUDA device");
    TORCH_CHECK(img2.device().is_cuda(), "Tensor img2 must be on CUDA device");
    TORCH_CHECK(img1.get_device() == img2.get_device(), "Input tensors must be on the same device");
    TORCH_CHECK(img1.dtype() == torch::kFloat32, "Only float32 is supported");
    TORCH_CHECK(img2.dtype() == torch::kFloat32, "Only float32 is supported");

    const at::cuda::OptionalCUDAGuard device_guard(device_of(img1));
    const int batch  = img1.size(0);
    const int channels = img1.size(1);
    const int height  = img1.size(2);
    const int width  = img1.size(3);

    const dim3 block_dim(BLOCK_X, BLOCK_Y);
    const dim3 grid_dim = calc_grid_dim(batch, height, width);

    const torch::Tensor img1_contiguous = img1.contiguous();
    const torch::Tensor img2_contiguous = img2.contiguous();

    torch::Tensor ssim_map = torch::zeros_like(img1_contiguous);

    auto create_optional_derivative_tensor = [&]() -> torch::Tensor {
        return train ? torch::zeros_like(img1_contiguous) : torch::empty({0}, img1_contiguous.options());
    };

    torch::Tensor dm_dmu1 = create_optional_derivative_tensor();
    torch::Tensor dm_dsigma1_sq = create_optional_derivative_tensor();
    torch::Tensor dm_dsigma12 = create_optional_derivative_tensor();

    auto get_conditional_ptr = [&](const torch::Tensor& t) -> float* {
        return train ? t.data_ptr<float>() : nullptr;
    };

    ssim_kernel<<<grid_dim, block_dim>>>(
        channels,
        height,
        width,
        C1,
        C2,
        img1_contiguous.data_ptr<float>(),
        img2_contiguous.data_ptr<float>(),
        ssim_map.data_ptr<float>(),
        get_conditional_ptr(dm_dmu1),
        get_conditional_ptr(dm_dsigma1_sq),
        get_conditional_ptr(dm_dsigma12)
    );

    const hipError_t err = hipDeviceSynchronize();
    TORCH_CHECK(err == hipSuccess, "CUDA error: ", hipGetErrorString(err));

    return std::make_tuple(ssim_map, dm_dmu1, dm_dsigma1_sq, dm_dsigma12);
}


//-------------------------------------------------------------------------------------------


/**
 * @brief Backward Kernel: Fused SSIM.
 *
 * Backward pass for fused SSIM Map Calculation (CUDA Kernel): Apply chain rule to get dL/d(img1) from partial
 *    derivatives (dm_dmu1, dm_dsigma1_sq, dm_dsigma12) and dL/dmap (the gradient from above).
 *
 * @param[in] channels number of image channels.
 * @param[in] height image height.
 * @param[in] width image width.
 * @param[in] C1 C1-const.
 * @param[in] C2 C2-const.
 * @param[in] img1 The first image.
 * @param[in] img2 The second image.
 * @param[in] dL_dmap dL_dmap partial derivative image.
 * @param[in] dm_dmu1 dm_dmu1 partial derivative image.
 * @param[in] dm_dsigma1_sq dm_dsigma1_sq partial derivative image.
 * @param[in] dm_dsigma12 dm_dsigma12 partial derivative image.
 * @param[out] dL_dimg1 dL_dimg1 partial derivative image.
 */
__global__ void ssim_backward_kernel(const int channels,
                                     const int height,
                                     const int width,
                                     const float C1,
                                     const float C2,
                                     const float* __restrict__ img1,
                                     const float* __restrict__ img2,
                                     const float* __restrict__ dL_dmap,
                                     const float* __restrict__ dm_dmu1,
                                     const float* __restrict__ dm_dsigma1_sq,
                                     const float* __restrict__ dm_dsigma12,
                                     float* __restrict__ dL_dimg1) {
    const cg::thread_block block = cg::this_thread_block();
    const dim3 g_idx = block.group_index();
    const dim3 thread_idx = block.thread_index();
    const int b_idx = g_idx.z;
    const int pix_y = g_idx.y * BLOCK_Y + thread_idx.y;
    const int pix_x = g_idx.x * BLOCK_X + thread_idx.x;
    const int pix_idx = pix_y * width + pix_x;
    const int num_pix = height * width;

    // Shared memory for the fused data:
    // [0]: dm_dmu1*dL, [1]: dm_dsigma1_sq*dL, [2]: dm_dsigma12*dL
    __shared__ float s_data[3][SHARED_Y][SHARED_X];
    __shared__ float s_scratch[CONV_Y][CONV_X][3];

    for (int c = 0; c < channels; ++c) {
        float p1 = 0.0f, p2 = 0.0f;
        if (pix_x < width && pix_y < height) {
            p1 = get_pix_value(img1, b_idx, c, pix_y, pix_x, channels, height, width);
            p2 = get_pix_value(img2, b_idx, c, pix_y, pix_x, channels, height, width);
        }

        // (1) Load + fuse multiplication
        {
            const int start_y = g_idx.y * BLOCK_Y;
            const int start_x = g_idx.x * BLOCK_X;

            const int tid = thread_idx.y * blockDim.x + thread_idx.x;
            const int warp_id = tid / 32;
            const int lane_id = tid % 32;
            const int total_threads = BLOCK_X * BLOCK_Y;
            const int num_warps = (total_threads + 31) / 32;

            for (int row = warp_id; row < SHARED_Y; row += num_warps) {
                const int gy = start_y + row - HALO;
                for (int col = lane_id; col < SHARED_X; col += 32) {
                    const int gx = start_x + col - HALO;

                    const float chain = get_pix_value(dL_dmap,       b_idx, c, gy, gx, channels, height, width);
                    const float vmu   = get_pix_value(dm_dmu1,       b_idx, c, gy, gx, channels, height, width);
                    const float vs1   = get_pix_value(dm_dsigma1_sq, b_idx, c, gy, gx, channels, height, width);
                    const float vs12  = get_pix_value(dm_dsigma12,   b_idx, c, gy, gx, channels, height, width);

                    s_data[0][row][col] = vmu  * chain;
                    s_data[1][row][col] = vs1  * chain;
                    s_data[2][row][col] = vs12 * chain;
                }
            }
        }
        block.sync();

        // (2) Horizontal pass
        {
            const int ly = thread_idx.y;
            const int lx = thread_idx.x + HALO;

            for (int pass = 0; pass < 2; ++pass) {
                const int yy = ly + pass * BLOCK_Y;
                if (yy < CONV_Y) {
                    float accum0 = 0.0f, accum1 = 0.0f, accum2 = 0.0f;

                    #pragma unroll
                    for (int d = 1; d <= HALO; ++d) {
                        const float w = gauss_coefs[HALO - d];
                        const float left0  = s_data[0][yy][lx - d];
                        const float left1  = s_data[1][yy][lx - d];
                        const float left2  = s_data[2][yy][lx - d];

                        const float right0 = s_data[0][yy][lx + d];
                        const float right1 = s_data[1][yy][lx + d];
                        const float right2 = s_data[2][yy][lx + d];

                        accum0 += (left0 + right0) * w;
                        accum1 += (left1 + right1) * w;
                        accum2 += (left2 + right2) * w;
                    }
                    // center
                    {
                        const float wc = gauss_coefs[HALO];
                        const float c0 = s_data[0][yy][lx];
                        const float c1 = s_data[1][yy][lx];
                        const float c2 = s_data[2][yy][lx];
                        accum0 += c0 * wc;
                        accum1 += c1 * wc;
                        accum2 += c2 * wc;
                    }

                    s_scratch[yy][thread_idx.x][0] = accum0;
                    s_scratch[yy][thread_idx.x][1] = accum1;
                    s_scratch[yy][thread_idx.x][2] = accum2;
                }
            }
        }
        block.sync();

        // (3) Vertical pass -> finalize dL/d(img1)
        if ((pix_x < width) && (pix_y < height)) {
            const int ly = thread_idx.y + HALO;
            const int lx = thread_idx.x;

            float sum0 = 0.0f, sum1 = 0.0f, sum2 = 0.0f;

            #pragma unroll
            for (int d = 1; d <= HALO; ++d) {
                const float w = gauss_coefs[HALO - d];
                const float* top = s_scratch[ly - d][lx];
                const float* bot = s_scratch[ly + d][lx];

                sum0 += (top[0] + bot[0]) * w;
                sum1 += (top[1] + bot[1]) * w;
                sum2 += (top[2] + bot[2]) * w;
            }
            // center
            {
                const float wc = gauss_coefs[HALO];
                const float* ctr = s_scratch[ly][lx];
                sum0 += ctr[0] * wc;
                sum1 += ctr[1] * wc;
                sum2 += ctr[2] * wc;
            }

            // final accumulation
            const float dL_dpix = sum0 + (2.0f * p1) * sum1 + (p2) * sum2;

            const int out_idx = b_idx * channels * num_pix + c * num_pix + pix_idx;
            dL_dimg1[out_idx] = dL_dpix;
        }
        block.sync();
    }
}

/**
 * @brief PyTorch Interface (Backward).
 *
 * PyTorch Interface for SSIM Map calculation (Backward pass)
 *   Takes the gradient wrt the SSIM map and
 *   the partial derivatives from forward;
 *   returns dL/d(img1).
 *
 * @param[in] C1 C1-const.
 * @param[in] C2 C2-const.
 * @param[in] img1 The first image.
 * @param[in] img2 The second image.
 * @param[in] dL_dmap dL_dmap partial derivative image.
 * @param[in] dm_dmu1 dm_dmu1 partial derivative image.
 * @param[in] dm_dsigma1_sq dm_dsigma1_sq partial derivative image.
 * @param[in] dm_dsigma12 dm_dsigma12 partial derivative image.
 * @return dL_dimg1 partial derivative image.
 */
torch::Tensor ssim_backward_cuda(const float C1,
                                 const float C2,
                                 const torch::Tensor& img1,
                                 const torch::Tensor& img2,
                                 const torch::Tensor& dL_dmap,
                                 const torch::Tensor& dm_dmu1,
                                 const torch::Tensor& dm_dsigma1_sq,
                                 const torch::Tensor& dm_dsigma12) {
    const at::cuda::OptionalCUDAGuard device_guard(device_of(img1));
    const int batch  = img1.size(0);
    const int channels = img1.size(1);
    const int height  = img1.size(2);
    const int width  = img1.size(3);

    const dim3 block_dim(BLOCK_X, BLOCK_Y);
    const dim3 grid_dim = calc_grid_dim(batch, height, width);

    const torch::Tensor img1_contiguous = img1.contiguous();
    const torch::Tensor img2_contiguous = img2.contiguous();

    torch::Tensor dL_dimg1 = torch::zeros_like(img1_contiguous);

    ssim_backward_kernel<<<grid_dim, block_dim>>>(
        channels,
        height,
        width,
        C1,
        C2,
        img1_contiguous.data_ptr<float>(),
        img2_contiguous.data_ptr<float>(),
        dL_dmap.contiguous().data_ptr<float>(),
        dm_dmu1.contiguous().data_ptr<float>(),
        dm_dsigma1_sq.contiguous().data_ptr<float>(),
        dm_dsigma12.contiguous().data_ptr<float>(),
        dL_dimg1.data_ptr<float>()
    );

    const hipError_t err = hipDeviceSynchronize();
    TORCH_CHECK(err == hipSuccess, "CUDA error: ", hipGetErrorString(err));

    return dL_dimg1;
}
